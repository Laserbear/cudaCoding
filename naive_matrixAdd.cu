#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cassert>



hipError_t addVecWithCuda(int *c,  int *a,  int *b, const unsigned int size);

hipError_t addMatrixWithCuda(int** c, int** a, int** b, const unsigned int rows, const unsigned int cols);

__global__ void addVecKernel(int* c, int* a, int* b, const unsigned int size)
{
    unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

__global__ void addMatrixKernel(int** c, int** a, int** b, const unsigned int rows, const unsigned int cols)
{
    unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < cols * rows) {
        c[i / cols][i % rows] = a[i / cols][i % rows] + b[i / cols][i % rows];
    }
}

void genRandVec(int* v, const unsigned int arraySize) {
    for (unsigned int i = 0; i < arraySize; i++) {
        v[i] = rand() % 100;
    }
}

void genRandMatrix(int** M, const unsigned int n, const unsigned int m) {
    for (unsigned int i = 0; i < n; i++) {
        genRandVec(M[i], m);
    }
}

void printVec(int* v, const unsigned int arraySize) {
    for (unsigned int i = 0; i < arraySize; i++) {
        std::cout << v[i] << " ";
    }
    //std::cout << std::endl;
}

void errorCheckVecAdd(int* a, int* b, int* c, const unsigned int arraySize) {
    for (unsigned int i = 0; i < arraySize; i++) {
        assert(c[i] == a[i] + b[i]);
    }
}

void errorCheckMatrixAdd(int** a, int** b, int** c, const unsigned int rows, const unsigned int cols) {
    for (unsigned int i = 0; i < cols; i++) {
        for (unsigned int j = 0; j < rows; j++) {
            assert(c[j][i] == a[j][i] + b[j][i]);
        }
    }
}

int main() 
{
    const unsigned int COLS = 50;
    const unsigned int ROWS = 25;
    int** a;
    int** b;
    genRandMatrix(a, ROWS, COLS);
    genRandMatrix(b, ROWS, COLS);
    int** c;

    // Add vectors in parallel.
    hipError_t cudaStatus = addMatrixWithCuda(c, a, b, ROWS, COLS);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    //printVec(a, arraySize); std::cout << "+ "; printVec(b, arraySize); std::cout << "= "; printVec(c, arraySize);
    
    //copy(&b[0], &b[arraySize], std::ostream_iterator<std::string>(std::cout, " "));
    //copy(&c[0], &c[arraySize], std::ostream_iterator<std::string>(std::cout, " "));
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    errorCheckMatrixAdd(a, b, c, ROWS, COLS);
    printf("Successful with %d by %d Matrix! of length", ROWS, COLS);
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addMatrixWithCuda(int **c, int **a, int **b, const unsigned int rows, const unsigned int cols)
//should actually be the same as vector add and just convert 2d array to 1d array
{
    int ** dev_a;
    int ** dev_b;
    int ** dev_c;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, rows * cols * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, rows * cols * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, rows * cols * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, rows * cols * sizeof(unsigned int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, rows * cols * sizeof(unsigned int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addMatrixKernel << < cols, rows>> > (dev_c, dev_a, dev_b, rows, cols);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, rows * cols * sizeof(unsigned int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
