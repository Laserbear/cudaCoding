﻿
#include "hip/hip_runtime.h"



#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cassert>



hipError_t addVecWithCuda(int *c,  int *a,  int *b, const unsigned int size);

hipError_t addMatrixWithCuda(int** c, int** a, int** b, const unsigned int rows, const unsigned int cols);

__global__ void addVecKernel(int* c, int* a, int* b, const unsigned int size)
{
    unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

__global__ void addMatrixKernel(int** c, int** a, int** b, const unsigned int rows, const unsigned int cols)
{
    unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < cols * rows) {
        c[i / rows][i % cols] = a[i / rows][i % cols] + b[i / rows][i % cols];
    }
}

void genRandVec(int* v, const unsigned int arraySize) {
    for (unsigned int i = 0; i < arraySize; i++) {
        v[i] = rand() % 100;
    }
}

void genRandMatrix(int** M, const unsigned int n, const unsigned int m) {
    for (unsigned int i = 0; i < m; i++) {
        genRandVec(M[i], n);
    }
}

void printVec(int* v, const unsigned int arraySize) {
    for (unsigned int i = 0; i < arraySize; i++) {
        std::cout << v[i] << " ";
    }
    //std::cout << std::endl;
}

void errorCheckVecAdd(int* a, int* b, int* c, const unsigned int arraySize) {
    for (unsigned int i = 0; i < arraySize; i++) {
        assert(c[i] == a[i] + b[i]);
    }
}

void errorCheckMatrixAdd(int** a, int** b, int** c, const unsigned int rows, const unsigned int cols) {
    for (unsigned int i = 0; i < cols; i++) {
        for (unsigned int j = 0; j < rows; j++) {
            c[j][i] = a[j][i] + b[j][i];
        }
    }
}

int main() 
{
    const unsigned int arraySize = 500;
    int a[arraySize] = {};
    int b[arraySize];
    genRandVec(a, arraySize);
    genRandVec(b, arraySize);
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addVecWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    //printVec(a, arraySize); std::cout << "+ "; printVec(b, arraySize); std::cout << "= "; printVec(c, arraySize);
    
    //copy(&b[0], &b[arraySize], std::ostream_iterator<std::string>(std::cout, " "));
    //copy(&c[0], &c[arraySize], std::ostream_iterator<std::string>(std::cout, " "));
    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    errorCheckVecAdd(a, b, c, arraySize);
    printf("Successful! with vectors of length %d", arraySize);
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, int *a, int *b, const unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addVecKernel<<<1, size>>>(dev_c, dev_a, dev_b, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
