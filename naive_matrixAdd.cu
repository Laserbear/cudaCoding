#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cassert>



hipError_t addVecWithCuda(int *c,  int *a,  int *b, const unsigned int size);

__global__ void addVecKernel(int* c, int* a, int* b, const unsigned int size)
{
    unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

void genRandVec(int* v, const unsigned int arraySize) {
    for (unsigned int i = 0; i < arraySize; i++) {
        v[i] = rand() % 100;
    }
}

void genRandMatrix(int** M, const unsigned int n, const unsigned int m) {
    for (unsigned int i = 0; i < n; i++) {
        M[i] = new int[m];
        genRandVec(M[i], m);
    }
}

void printVec(int* v, const unsigned int arraySize) {
    for (unsigned int i = 0; i < arraySize; i++) {
        std::cout << v[i] << " ";
    }
    //std::cout << std::endl;
}

void errorCheckVecAdd(int* a, int* b, int* c, const unsigned int arraySize) {
    for (unsigned int i = 0; i < arraySize; i++) {
        assert(c[i] == a[i] + b[i]);
    }
}

void errorCheckMatrixAdd(int** a, int** b, int** c, const unsigned int rows, const unsigned int cols) {
    for (unsigned int i = 0; i < cols; i++) {
        for (unsigned int j = 0; j < rows; j++) {
            assert(c[j][i] == a[j][i] + b[j][i]);
        }
    }
}

//Take a matrix m and flatten it into a flat vector f
void flattenMatrix(int* f, int** m, const unsigned int rows, const unsigned int cols) {
    for (unsigned int i = 0; i < rows; i++) {
        for (unsigned int j = 0; i < cols; j++) {
            f[i * cols + j] = m[i][j];
        }
    }
}

void unFlattenMatrix(int** m, int* f, const unsigned int rows, const unsigned int cols) {
    for (unsigned int i = 0; i < rows; i++) {
        for (unsigned int j = 0; i < cols; j++) {
            m[i][j] = f[i * cols + j];
        }
    }
}

int main()
{
    const unsigned int COLS = 5;
    const unsigned int ROWS = 5;
    int* a[ROWS];
    int* b[ROWS];
    genRandMatrix(a, ROWS, COLS);
    genRandMatrix(b, ROWS, COLS);
    printf("Matrices generated successfully");
    int* c[ROWS];

    //Flatten matrices
    int a_f[ROWS * COLS];
    int b_f[ROWS * COLS];
    int c_f[ROWS * COLS];

    flattenMatrix(a_f, a, ROWS, COLS);
    flattenMatrix(b_f, b, ROWS, COLS);
    // Add vectors in parallel.
    printf("Vectors flattened successfully");
    hipError_t cudaStatus = addVecWithCuda(c_f, a_f, b_f, ROWS * COLS);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    //printVec(a, arraySize); std::cout << "+ "; printVec(b, arraySize); std::cout << "= "; printVec(c, arraySize);
    
    //copy(&b[0], &b[arraySize], std::ostream_iterator<std::string>(std::cout, " "));
    //copy(&c[0], &c[arraySize], std::ostream_iterator<std::string>(std::cout, " "));
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    unFlattenMatrix(c, c_f, ROWS, COLS);
    //errorCheckMatrixAdd(a, b, c, ROWS, COLS);
    printf("Successful with %d by %d Matrix! of length", ROWS, COLS);
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addVecWithCuda(int *c, int *a, int *b, const unsigned int size)
//should actually be the same as vector add and just convert 2d array to 1d array to save GMEM accesses
{
    
    int * dev_a;
    int * dev_b;
    int * dev_c;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof( int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof( int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addVecKernel << < 1, size >> > (dev_c, dev_a, dev_b, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof( int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
